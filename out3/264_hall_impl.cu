#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation. All rights reserved.
// Licensed under the MIT License.

#include "orttraining/training_ops/rocm/reduction/all_impl.h"

#include <thrust/logical.h>
#include <thrust/functional.h>
#include <thrust/execution_policy.h>

#ifdef _WIN32
#pragma warning(disable : 4244)
#endif
namespace onnxruntime {
namespace rocm {

__global__ void assign_true(bool* ptr) {
  *ptr = true;
}

__global__ void assign_false(bool* ptr) {
  *ptr = false;
}

template<>
void LaunchAllKernel(hipStream_t stream, const bool* data, const int size, bool* output) {
  if(thrust::all_of(thrust::hip::par.on(stream), data, data + size, thrust::identity<bool>())) {
    assign_true<<<1, 1, 0, stream>>>(output);
  }
  else
  {
    assign_false<<<1, 1, 0, stream>>>(output);
  }
}

}  // namespace rocm
}  // namespace onnxruntime
